// $ nvcc --expt-extended-lambda -std=c++14 -I. chaining_executor_demo.cu
#include <iostream>
#include <typeinfo>
#include <cassert>
#include "chaining_executor.hpp"
#include "submit.hpp"
#include "cuda/single_executor.hpp"


struct my_receiver
{
  template<class T>
  __host__ __device__
  void set_value(T arg)
  {
#ifndef __CUDA_ARCH__
    std::cout << "my_receiver::set_value: received " << typeid(arg).name() << std::endl;
#else
    printf("Hello world from my_receiver\n");
#endif
  }
};

int main()
{
  chaining_executor<cuda::single_executor> ex = make_chaining_executor(cuda::single_executor());

  just<chaining_executor<cuda::single_executor>> s1 = ex.schedule();

  op::submit(s1, my_receiver());

  auto s2 = ex.make_value_task(std::move(s1), [] __host__ __device__ (chaining_executor<cuda::single_executor>)
  {
    printf("Hello world from value task\n");
    return 0;
  });

  op::submit(std::move(s2), my_receiver());

  if(hipError_t error = hipDeviceSynchronize())
  {
    throw std::runtime_error("CUDA error after hipDeviceSynchronize: " + std::string(hipGetErrorString(error)));
  }

  return 0; 
}

