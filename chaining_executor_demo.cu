// $ nvcc --expt-extended-lambda -std=c++14 -I../agency-tot chaining_executor_demo.cu
#include <iostream>
#include <typeinfo>
#include <cassert>
#include "chaining_executor.hpp"


struct my_receiver
{
  template<class T>
  __host__ __device__
  void set_value(T arg)
  {
#ifndef __CUDA_ARCH__
    std::cout << "my_receiver::set_value: received " << typeid(arg).name() << std::endl;
#else
    printf("Hello world from my_receiver\n");
#endif
  }
};

int main()
{
  chaining_executor ex;

  just<chaining_executor> s1 = ex.schedule();

  s1.submit(my_receiver());

  auto s2 = ex.make_value_task(std::move(s1), [] __host__ __device__ (chaining_executor)
  {
    printf("Hello world from value task\n");
    return 0;
  });

  std::move(s2).submit(my_receiver());

  if(hipError_t error = hipDeviceSynchronize())
  {
    throw std::runtime_error("CUDA error after hipDeviceSynchronize: " + std::string(hipGetErrorString(error)));
  }

  return 0; 
}

