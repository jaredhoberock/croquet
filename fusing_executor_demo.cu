// $ nvcc --expt-extended-lambda -std=c++14 -I../agency-tot fusing_executor_demo.cu
#include <iostream>
#include <typeinfo>
#include "fusing_executor.hpp"

struct my_receiver
{
  template<class T>
  __host__ __device__
  void set_value(T arg)
  {
#ifndef __CUDA_ARCH__
    std::cout << "my_receiver::set_value: received " << typeid(arg).name() << std::endl;
#else
    printf("Hello world from my_receiver\n");
#endif
  }
};

int main()
{
  fusing_executor ex;

  just<fusing_executor> s1 = ex.schedule();

  s1.submit(my_receiver());

  auto s2 = ex.make_value_task(std::move(s1), [] __host__ __device__ (fusing_executor)
  {
    printf("Hello world from value task\n");
    return 0;
  });

  std::move(s2).submit(my_receiver());

  if(hipError_t error = hipDeviceSynchronize())
  {
    throw std::runtime_error("CUDA error after hipDeviceSynchronize: " + std::string(hipGetErrorString(error)));
  }

  return 0; 
}

